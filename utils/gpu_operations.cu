#include <hip/hip_runtime.h>

#include <cstdio>

#include "gpu_operations.hpp"
#include "constants.hpp"

// Error checking macro
#define CUDA_CHECK_ERROR() {                                          \
    hipError_t err = hipGetLastError();                             \
    if (err != hipSuccess) {                                         \
        printf("CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__);  \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
    printf("CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__);  \
}

/**
 * @brief Prepares GPU memory space and copies initial data from host to device.
 *
 * @param d_ALGEBRAIC Pointer to device memory for algebraic variables.
 * @param ORd_num_of_algebraic Number of algebraic variables.
 * @param sample_size Number of samples.
 * @param d_CONSTANTS Pointer to device memory for constants.
 * @param ORd_num_of_constants Number of constants.
 * @param d_RATES Pointer to device memory for rates.
 * @param ORd_num_of_rates Number of rates.
 * @param d_STATES Pointer to device memory for states.
 * @param ORd_num_of_states Number of states.
 * @param d_p_param Pointer to device memory for parameters.
 * @param temp_result Pointer to device memory for temporary results.
 * @param cipa_result Pointer to device memory for CIPA results.
 * @param d_STATES_RESULT Pointer to device memory for states results.
 * @param d_ic50 Pointer to device memory for IC50 data.
 * @param ic50 Pointer to host memory for IC50 data.
 * @param d_conc Pointer to device memory for concentration data.
 * @param conc Pointer to host memory for concentration data.
 * @param p_param Pointer to host memory for parameters.
 */
void prepingGPUMemory(int sample_size, double *&d_ALGEBRAIC, double *&d_CONSTANTS, double *&d_RATES, double *&d_STATES,
                      double *&d_mec_ALGEBRAIC, double *&d_mec_CONSTANTS, double *&d_mec_RATES, double *&d_mec_STATES,
                      param_t *&d_p_param, cipa_t *&temp_result, cipa_t *&cipa_result, double *&d_STATES_RESULT, double *&d_ic50,           
                      double *ic50, double *&d_conc, double *conc, param_t *p_param, double *&d_herg, double *herg) {
    printf("preparing GPU memory space \n");

    // Allocate memory on the device
    hipMalloc(&d_ALGEBRAIC, ORd_num_of_algebraic * sample_size * sizeof(double));
    hipMalloc(&d_CONSTANTS, ORd_num_of_constants * sample_size * sizeof(double));
    hipMalloc(&d_RATES, ORd_num_of_rates * sample_size * sizeof(double));
    hipMalloc(&d_STATES, ORd_num_of_states * sample_size * sizeof(double));
    hipMalloc(&d_mec_ALGEBRAIC, Land_num_of_algebraic * sample_size * sizeof(double));
    hipMalloc(&d_mec_CONSTANTS, Land_num_of_constants * sample_size * sizeof(double));
    hipMalloc(&d_mec_RATES, Land_num_of_rates * sample_size * sizeof(double));
    hipMalloc(&d_mec_STATES, Land_num_of_states * sample_size * sizeof(double));
    hipMalloc(&d_p_param, sizeof(param_t));
    hipMalloc(&temp_result, sample_size * sizeof(cipa_t));
    hipMalloc(&cipa_result, sample_size * sizeof(cipa_t));
    hipMalloc(&d_STATES_RESULT, ORd_num_of_states * sample_size * sizeof(double));

    // Allocate memory for IC50 and concentration data
    hipMalloc(&d_ic50, sample_size * 14 * sizeof(double));
    hipMalloc(&d_conc, sample_size * sizeof(double));
    hipMalloc(&d_herg, 6 * sizeof(double));

    // Copy data from host to device
    printf("Copying sample files to GPU memory space \n");
    hipMemcpy(d_herg, herg, 6 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_ic50, ic50, sample_size * 14 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_conc, conc, sample_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_p_param, p_param, sizeof(param_t), hipMemcpyHostToDevice);
}

void prepingGPUMemoryPostpro(int sample_size, double *&d_ALGEBRAIC, double *&d_CONSTANTS, double *&d_RATES, double *&d_STATES, double *d_STATES_cache,
                      double *&d_mec_ALGEBRAIC, double *&d_mec_CONSTANTS, double *&d_mec_RATES, double *&d_mec_STATES,
                      param_t *&d_p_param, cipa_t *&temp_result, cipa_t *&cipa_result, double *&d_STATES_RESULT, double *&d_ic50,
                      double *ic50, double *&d_cvar, double *cvar, double *&d_conc, double *conc, double *&d_herg, double *herg, param_t *p_param, double *cache,
                      double *time, double *dt, double *states, double *ical, double *inal, double *cai_result, double *ina, double *ito, double *ikr, double *iks, double *ik1, double *tension) {
    
    printf("preparing GPU memory space \n");
    // Allocate memory on the device
    // hipMalloc(&d_ALGEBRAIC, ORd_num_of_algebraic * sample_size * sizeof(double));
    // hipMalloc(&d_CONSTANTS, ORd_num_of_constants * sample_size * sizeof(double));
    // hipMalloc(&d_RATES, ORd_num_of_rates * sample_size * sizeof(double));
    // hipMalloc(&d_STATES, ORd_num_of_states * sample_size * sizeof(double));
    // hipMalloc(&d_STATES_cache, (ORd_num_of_states+2) * sample_size * sizeof(double));
    CUDA_CHECK_ERROR(); 

    hipMalloc(&d_mec_ALGEBRAIC, Land_num_of_algebraic * sample_size * sizeof(double));
    hipMalloc(&d_mec_CONSTANTS, Land_num_of_constants * sample_size * sizeof(double));
    hipMalloc(&d_mec_RATES, Land_num_of_rates * sample_size * sizeof(double));
    hipMalloc(&d_mec_STATES, Land_num_of_states * sample_size * sizeof(double));

    hipMalloc(&d_p_param, sizeof(param_t));

    hipMalloc(&temp_result, sample_size * sizeof(cipa_t));
    hipMalloc(&cipa_result, sample_size * sizeof(cipa_t));
    

    hipMalloc(&time, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&dt, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&states, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ical, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&inal, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&cai_result, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ina, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ito, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ikr, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&iks, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ik1, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&tension, sample_size * datapoint_size * sizeof(double));
    // hipMalloc(&d_STATES_RESULT, ORd_num_of_states * sample_size * sizeof(double)); // used by in silico, here, its just for the function

    printf("Copying sample files to GPU memory space \n");
    hipMalloc(&d_ic50, sample_size * 14 * sizeof(double));
    hipMalloc(&d_cvar, sample_size * 18 * sizeof(double));
    hipMalloc(&d_conc, sample_size * sizeof(double));
    hipMalloc(&d_herg, 6 * sizeof(double));
    
    // hipMemcpy(d_STATES_cache, cache, (ORd_num_of_states+2) * sample_size * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();      
    hipMemcpy(d_ic50, ic50, sample_size * 14 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_cvar, cvar, sample_size * 18 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_conc, conc, sample_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_p_param, p_param, sizeof(param_t), hipMemcpyHostToDevice);
    hipMemcpy(d_herg, herg, 6 * sizeof(double), hipMemcpyHostToDevice);

}

/**
 * @brief Frees allocated memory on both the host and device.
 *
 * @param d_ALGEBRAIC Pointer to device memory for algebraic variables.
 * @param d_CONSTANTS Pointer to device memory for constants.
 * @param d_RATES Pointer to device memory for rates.
 * @param d_STATES Pointer to device memory for states.
 * @param d_p_param Pointer to device memory for parameters.
 * @param temp_result Pointer to device memory for temporary results.
 * @param cipa_result Pointer to device memory for CIPA results.
 * @param d_STATES_RESULT Pointer to device memory for states results.
 * @param d_ic50 Pointer to device memory for IC50 data.
 * @param ic50 Pointer to host memory for IC50 data.
 * @param conc Pointer to host memory for concentration data.
 * @param h_states Pointer to host memory for states.
 * @param h_cipa_result Pointer to host memory for CIPA results.
 * @param p_param Pointer to host memory for parameters.
 */
void freeingMemory(double *d_ALGEBRAIC, double *d_CONSTANTS, double *d_RATES, double *d_STATES, double *d_mec_ALGEBRAIC,
                   double *d_mec_CONSTANTS, double *d_mec_RATES, double *d_mec_STATES, param_t *d_p_param,
                   cipa_t *temp_result, cipa_t *cipa_result, double *d_STATES_RESULT, double *d_ic50, double *ic50,
                   double *conc, double *h_states, cipa_t *h_cipa_result, param_t *p_param) {
    // Free GPU memory
    hipFree(d_ALGEBRAIC);
    hipFree(d_CONSTANTS);
    hipFree(d_RATES);
    hipFree(d_STATES);
    hipFree(d_mec_ALGEBRAIC);
    hipFree(d_mec_CONSTANTS);
    hipFree(d_mec_RATES);
    hipFree(d_mec_STATES);
    hipFree(d_p_param);
    hipFree(temp_result);
    hipFree(cipa_result);
    hipFree(d_STATES_RESULT);
    hipFree(d_ic50);

    // Free CPU memory
    free(ic50);
    free(conc);
    free(h_states);
    free(h_cipa_result);
    delete p_param;
}

/**
 * @brief Checks the available GPU memory.
 *
 * @param datasize Size of the data to be checked against available GPU memory.
 * @return int 0 if successful, 1 if insufficient memory.
 */
int gpu_check(unsigned int datasize) {
    int num_gpus;
    float percent;
    int id;
    size_t free, total;
    hipGetDeviceCount(&num_gpus);
    for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
        hipSetDevice(gpu_id);
        hipGetDevice(&id);
        hipMemGetInfo(&free, &total);
        percent = (free / (float)total);
        printf("GPU No %d\nFree Memory: %ld, Total Memory: %ld (%f percent free)\n", id, free, total, percent * 100.0);
    }
    percent = 1.0 - (datasize / (float)total);

    return (percent >= 0) ? 0 : 1;
}
